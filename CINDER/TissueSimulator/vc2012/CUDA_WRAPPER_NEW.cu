#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>

#include "CUDA_WRAPPER.h"
#include "CUDA_GLOBALS.h"

#define IDX (blockIdx.x * blockDim.x + threadIdx.x)
#define LOCAL_IDX threadIdx.x

__shared__ float F[NUM_THREADS_PER_BLOCK][3][3];

__device__ float sqr(float x)
{
	return x * x;
}

__device__ float determinantF(int idx)
{
	return F[idx][0][0]
		* (F[idx][1][1] * F[idx][2][2] - F[idx][1][2] * F[idx][2][1])
		- F[idx][0][1]
		* (F[idx][1][0] * F[idx][2][2] - F[idx][1][2] * F[idx][2][0])
		+ F[idx][0][2]
		* (F[idx][1][0] * F[idx][2][1] - F[idx][1][1] * F[idx][2][0]);
}

__device__ void calculateF(int globalIdx, int idx, float* positions, float* refShapeMatrixInverse,
	int* indices, int trueNumConstraints, int numParticles)
{
	int localIndices[4];
	localIndices[0] = indices[globalIdx + trueNumConstraints * 0] * 3;
	localIndices[1] = indices[globalIdx + trueNumConstraints * 1] * 3;
	localIndices[2] = indices[globalIdx + trueNumConstraints * 2] * 3;
	localIndices[3] = indices[globalIdx + trueNumConstraints * 3] * 3;
	//localIndices[0] = indices[globalIdx + trueNumConstraints * 0];
	//localIndices[1] = indices[globalIdx + trueNumConstraints * 1];
	//localIndices[2] = indices[globalIdx + trueNumConstraints * 2];
	//localIndices[3] = indices[globalIdx + trueNumConstraints * 3];

	float temp[3][3];

	//float node3[3];
	//node3[0] = positions[localIndices[3] + 0];
	//node3[1] = positions[localIndices[3] + 1];
	//node3[2] = positions[localIndices[3] + 2];

	//1. Calculate Deformed Shape Matrix
	temp[0][0] = positions[localIndices[0] + 0] - positions[localIndices[3] + 0];
	temp[1][0] = positions[localIndices[0] + 1] - positions[localIndices[3] + 1];
	temp[2][0] = positions[localIndices[0] + 2] - positions[localIndices[3] + 2];
	temp[0][1] = positions[localIndices[1] + 0] - positions[localIndices[3] + 0];
	temp[1][1] = positions[localIndices[1] + 1] - positions[localIndices[3] + 1];
	temp[2][1] = positions[localIndices[1] + 2] - positions[localIndices[3] + 2];
	temp[0][2] = positions[localIndices[2] + 0] - positions[localIndices[3] + 0];
	temp[1][2] = positions[localIndices[2] + 1] - positions[localIndices[3] + 1];
	temp[2][2] = positions[localIndices[2] + 2] - positions[localIndices[3] + 2];

	//2. Multiply 
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += temp[row][i] * refShapeMatrixInverse[(i * 3 + col) * trueNumConstraints + globalIdx];
			}

			F[idx][row][col] = sum;
		}
	}
}


__device__ __forceinline__ float calculateStrainEnergy_NEO_HOOKEAN(float volume, float lambda, float mu, float I1, float I3)
{
	return volume * (0.5f * mu * (I1 - log(I3) - 3.0f) + (lambda / 8.0f) * (log(I3) * log(I3)));
}

__device__ void updatePositions_recomputeGradients(int globalIdx, int idx, float lagrangeMultiplier, float* positions,
	float* masses, int* indices, int trueNumConstraints, int numParticles, float volume, float* refShapeMatrixInverse,
	float I3, float lambda, float mu, float det,
	float* globalU, float* globalV)
{
	//1. Copy refShapeMatrixInverse from global memory
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		//We need the TRANSPOSE of the reference shape matrix inverse
	//		Gradient[idx][row][col] = refShapeMatrixInverse[(col * 3 + row) * trueNumConstraints + globalIdx];
	//	}
	//}

	float temp0[3][3];
	float temp[3][3];
	// Load U
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			temp0[row][col] = globalU[IDX * 9 + row * 3 + col];
		}
	}

	//printf("%.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n",
	//	temp0[0][0], temp0[0][1], temp0[0][2],
	//	temp0[1][0], temp0[1][1], temp0[1][2],
	//	temp0[2][0], temp0[2][1], temp0[2][2]);

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;
			for (int i = 0; i < 3; ++i)
			{
				float FInverseTEntry = 0.0f;

				if (F[LOCAL_IDX][i][col] != 0.0f && i == col)
				{
					FInverseTEntry = 1.0f / F[LOCAL_IDX][i][col];
				}

				//FInverseTEntry /= det;

				sum += temp0[row][i] * (F[LOCAL_IDX][i][col] * mu - (FInverseTEntry * mu) + FInverseTEntry * ((lambda * log(I3)) / 2.0f));
			}
			temp[row][col] = sum;
		}
	}

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;
			for (int i = 0; i < 3; ++i)
			{
				sum += temp[row][i] * globalV[IDX * 9 + col * 3 + i];
			}
			temp0[row][col] = sum;
		}
	}

	//3. Multiply with First Piola-Kirchoff Stress tensor
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += temp0[row][i] * refShapeMatrixInverse[(col * 3 + i) * trueNumConstraints + globalIdx];
			}

			temp[row][col] = sum;
		}
	}

	int localIndices[4];
	localIndices[0] = indices[globalIdx + trueNumConstraints * 0] * 3;
	localIndices[1] = indices[globalIdx + trueNumConstraints * 1] * 3;
	localIndices[2] = indices[globalIdx + trueNumConstraints * 2] * 3;
	localIndices[3] = indices[globalIdx + trueNumConstraints * 3] * 3;

	for (int i = 0; i < 3; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			atomicAdd(&positions[localIndices[i] + j], masses[globalIdx + i * trueNumConstraints] * lagrangeMultiplier * temp[j][i] * volume);
		}
	}

	for (int j = 0; j < 3; ++j)
	{
		float sum = 0.0f;
		for (int col = 0; col < 3; ++col)
		{
			sum += temp[j][col] * volume;
		}

		atomicAdd(&positions[localIndices[3] + j], masses[globalIdx + 3 * trueNumConstraints] * lagrangeMultiplier * -sum);
	}
}


__device__ __forceinline__ float calculateTraceFTransposeF_INPLACE()
{
	float trace = 0.0f;
	for (int diagIdx = 0; diagIdx < 3; ++diagIdx)
	{
		for (int i = 0; i < 3; ++i)
		{
			trace += F[threadIdx.x][i][diagIdx] * F[threadIdx.x][i][diagIdx];
		}
	}

	return trace;
}

__device__ __forceinline__ float getFtFEntry(int row, int col)
{
	return F[threadIdx.x][0][row] * F[threadIdx.x][0][col]
		+ F[threadIdx.x][1][row] * F[threadIdx.x][1][col]
		+ F[threadIdx.x][2][row] * F[threadIdx.x][2][col];
}

__device__ __forceinline__ float calculatedeterminantFTransposeF_INPLACE()
{
	return (getFtFEntry(0, 0)
		* (getFtFEntry(1, 1) * getFtFEntry(2, 2) - getFtFEntry(1, 2) * getFtFEntry(2, 1)))
		- (getFtFEntry(0, 1)
		* (getFtFEntry(1, 0) * getFtFEntry(2, 2) - getFtFEntry(1, 2) * getFtFEntry(2, 0)))
		+ (getFtFEntry(0, 2)
		* (getFtFEntry(1, 0) * getFtFEntry(2, 1) - getFtFEntry(1, 1) * getFtFEntry(2, 0)));
}

__device__ void calculateStrainEnergyGradient_NEO_HOOKEAN_INPLACE(int globalIdx, int idx, float volume, float* refShapeMatrixInverse, int trueNumConstraints, float mu, float lambda, float I3,
	float& snGr0, float& snGr1, float& snGr2, float& snGr3, float det)
{
	//1. Copy refShapeMatrixInverse from global memory
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		//We need the TRANSPOSE of the reference shape matrix inverse
	//		Gradient[idx][row][col] = refShapeMatrixInverse[(col * 3 + row) * trueNumConstraints + globalIdx];
	//	}
	//}

	float temp[3][3];

	//3. Multiply with First Piola-Kirchoff Stress tensor
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				float FInverseTEntry;

				if (row == 0)
				{
					if (i == 0)
					{
						FInverseTEntry = F[idx][1][1] * F[idx][2][2] - F[idx][2][1] * F[idx][1][2];
					}
					else if (i == 1)
					{
						FInverseTEntry = -(F[idx][1][0] * F[idx][2][2] - F[idx][2][0] * F[idx][1][2]);
					}
					else if (i == 2)
					{
						FInverseTEntry = F[idx][1][0] * F[idx][2][1] - F[idx][2][0] * F[idx][1][1];
					}
				}
				else if (row == 1)
				{
					if (i == 0)
					{
						FInverseTEntry = -(F[idx][0][1] * F[idx][2][2] - F[idx][2][1] * F[idx][0][2]);
					}
					else if (i == 1)
					{
						FInverseTEntry = F[idx][0][0] * F[idx][2][2] - F[idx][2][0] * F[idx][0][2];
					}
					else if (i == 2)
					{
						FInverseTEntry = -(F[idx][0][0] * F[idx][2][1] - F[idx][2][0] * F[idx][0][1]);
					}
				}
				else if (row == 2)
				{
					if (i == 0)
					{
						FInverseTEntry = F[idx][0][1] * F[idx][1][2] - F[idx][1][1] * F[idx][0][2];
					}
					else if (i == 1)
					{
						FInverseTEntry = -(F[idx][0][0] * F[idx][1][2] - F[idx][1][0] * F[idx][0][2]);
					}
					else if (i == 2)
					{
						FInverseTEntry = F[idx][0][0] * F[idx][1][1] - F[idx][1][0] * F[idx][0][1];
					}
				}

				FInverseTEntry /= det;

				float PFEntry = F[idx][row][i] * mu - (FInverseTEntry * mu) + FInverseTEntry * ((lambda * log(I3)) / 2.0f);

				//correct PF entry to take into account diagonalisation


				sum += PFEntry * refShapeMatrixInverse[(col * 3 + i) * trueNumConstraints + globalIdx];
			}

			temp[row][col] = sum;
		}
	}

	//4. Copy back
	snGr0 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		snGr0 += sqr(temp[i][0] * volume);
	}
	snGr0 = sqrtf(snGr0);

	snGr1 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		snGr1 += sqr(temp[i][1] * volume);
	}
	snGr1 = sqrtf(snGr1);

	snGr2 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		snGr2 += sqr(temp[i][2] * volume);
	}
	snGr2 = sqrtf(snGr2);

	//4. Calculate last column
	snGr3 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		float sum = 0.0f;
		for (int col = 0; col < 3; ++col)
		{
			sum += temp[i][col] * volume;
		}
		snGr3 += sqr(sum);
	}
	snGr3 = sqrtf(snGr3);
}

__device__ void calculateStrainEnergyGradient_NEO_HOOKEAN_INPLACE(int globalIdx, int idx, float volume, float* refShapeMatrixInverse, int trueNumConstraints, float mu, float lambda, float I3,
	float& snGr0, float& snGr1, float& snGr2, float& snGr3, float det,
	float* globalU, float* globalV)
{
	//1. Copy refShapeMatrixInverse from global memory
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		//We need the TRANSPOSE of the reference shape matrix inverse
	//		Gradient[idx][row][col] = refShapeMatrixInverse[(col * 3 + row) * trueNumConstraints + globalIdx];
	//	}
	//}


	float temp0[3][3];
	float temp[3][3];
	// Load U
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			temp0[row][col] = globalU[IDX * 9 + row * 3 + col];
		}
	}

	//printf("%.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n",
	//	temp0[0][0], temp0[0][1], temp0[0][2],
	//	temp0[1][0], temp0[1][1], temp0[1][2],
	//	temp0[2][0], temp0[2][1], temp0[2][2]);

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;
			for (int i = 0; i < 3; ++i)
			{
				float FInverseTEntry = 0.0f;
				
				if (F[LOCAL_IDX][i][col] != 0.0f && i == col)
				{
					FInverseTEntry = 1.0f / F[LOCAL_IDX][i][col];
				}

				//FInverseTEntry /= det;

				sum += temp0[row][i] * (F[LOCAL_IDX][i][col] * mu - (FInverseTEntry * mu) + FInverseTEntry * ((lambda * log(I3)) / 2.0f));
			}
			temp[row][col] = sum;
		}
	}

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;
			for (int i = 0; i < 3; ++i)
			{
				sum += temp[row][i] * globalV[IDX * 9 + col * 3 + i];
			}
			temp0[row][col] = sum;
		}
	}

	//3. Multiply with First Piola-Kirchoff Stress tensor
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += temp0[row][i] * refShapeMatrixInverse[(col * 3 + i) * trueNumConstraints + globalIdx];
			}

			temp[row][col] = sum;
		}
	}

	//4. Copy back
	snGr0 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		snGr0 += sqr(temp[i][0] * volume);
	}
	snGr0 = sqrtf(snGr0);

	snGr1 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		snGr1 += sqr(temp[i][1] * volume);
	}
	snGr1 = sqrtf(snGr1);

	snGr2 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		snGr2 += sqr(temp[i][2] * volume);
	}
	snGr2 = sqrtf(snGr2);

	//4. Calculate last column
	snGr3 = 0.0f;
	for (int i = 0; i < 3; ++i)
	{
		float sum = 0.0f;
		for (int col = 0; col < 3; ++col)
		{
			sum += temp[i][col] * volume;
		}
		snGr3 += sqr(sum);
	}
	snGr3 = sqrtf(snGr3);
}

__global__ void solveFEMConstraint(float* positions, int* indices, float* inverseMass, float* volume, float* refShapeMatrixInverse,
	float lambda, float mu, int trueNumConstraints, int numParticles, float* globalF, float* globalU, float* globalV)
{
	if (IDX > trueNumConstraints)
	{
		return;
	}

	//1. Calculate Deformation Gradient F
	//calculateF(IDX, threadIdx.x, positions, refShapeMatrixInverse, indices, trueNumConstraints, numParticles);

	//1. Load Deformation Gradient
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			F[LOCAL_IDX][row][col] = 0.0f;
		}
	}

	F[LOCAL_IDX][0][0] = globalF[IDX * 3 + 0];
	F[LOCAL_IDX][1][1] = globalF[IDX * 3 + 1];
	F[LOCAL_IDX][2][2] = globalF[IDX * 3 + 2];

	//printf("%.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n",
	//	F[LOCAL_IDX][0][0], F[LOCAL_IDX][0][1], F[LOCAL_IDX][0][2],
	//	F[LOCAL_IDX][1][0], F[LOCAL_IDX][1][1], F[LOCAL_IDX][1][2],
	//	F[LOCAL_IDX][2][0], F[LOCAL_IDX][2][1], F[LOCAL_IDX][2][2]);

	//3. Compute Invariants

	float I3 = calculatedeterminantFTransposeF_INPLACE();
	float I1 = calculateTraceFTransposeF_INPLACE();

	//6. Calculate Strain Energy Gradient
	float snGr0;
	float snGr1;
	float snGr2;
	float snGr3;

	float det = determinantF(LOCAL_IDX);

	calculateStrainEnergyGradient_NEO_HOOKEAN_INPLACE(IDX, threadIdx.x, volume[IDX], refShapeMatrixInverse, trueNumConstraints, mu, lambda, I3,
		snGr0, snGr1, snGr2, snGr3, det,
		globalU, globalV);

	//7. Calculate Lagrange Multiplier
	float denominator = 0.0f;
	denominator += inverseMass[IDX + 0 * trueNumConstraints] * snGr0;
	denominator += inverseMass[IDX + 1 * trueNumConstraints] * snGr1;
	denominator += inverseMass[IDX + 2 * trueNumConstraints] * snGr2;
	denominator += inverseMass[IDX + 3 * trueNumConstraints] * snGr3;

	float lagrangeMultiplier = -(calculateStrainEnergy_NEO_HOOKEAN(volume[IDX], lambda, mu, I1, I3) / denominator);

	if (isnan(lagrangeMultiplier))
	{
		return;
	}

	//8. Update Positions
	updatePositions_recomputeGradients(IDX, threadIdx.x, lagrangeMultiplier, positions, inverseMass, indices, trueNumConstraints, numParticles,
		volume[IDX], refShapeMatrixInverse, I3, lambda, mu, det,
		globalU, globalV);
}

__global__ void computeDiagonalF(float* positions, int* indices, float* globalF, float* globalU, float* globalV, float* refShapeMatrixInverse, int trueNumConstraints)
{
	if (IDX > trueNumConstraints)
	{
		return;
	}

	float F_functionLevel[3][3];

	//1. COMPUTE F--------------------------------------------------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------------------------
	int localIndices[4];
	localIndices[0] = indices[IDX + trueNumConstraints * 0] * 3;
	localIndices[1] = indices[IDX + trueNumConstraints * 1] * 3;
	localIndices[2] = indices[IDX + trueNumConstraints * 2] * 3;
	localIndices[3] = indices[IDX + trueNumConstraints * 3] * 3;

	float temp[3][3];

	//1. Calculate Deformed Shape Matrix
	temp[0][0] = positions[localIndices[0] + 0] - positions[localIndices[3] + 0];
	temp[1][0] = positions[localIndices[0] + 1] - positions[localIndices[3] + 1];
	temp[2][0] = positions[localIndices[0] + 2] - positions[localIndices[3] + 2];
	temp[0][1] = positions[localIndices[1] + 0] - positions[localIndices[3] + 0];
	temp[1][1] = positions[localIndices[1] + 1] - positions[localIndices[3] + 1];
	temp[2][1] = positions[localIndices[1] + 2] - positions[localIndices[3] + 2];
	temp[0][2] = positions[localIndices[2] + 0] - positions[localIndices[3] + 0];
	temp[1][2] = positions[localIndices[2] + 1] - positions[localIndices[3] + 1];
	temp[2][2] = positions[localIndices[2] + 2] - positions[localIndices[3] + 2];

	//2. Multiply 
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += temp[row][i] * refShapeMatrixInverse[(i * 3 + col) * trueNumConstraints + IDX];
			}

			F_functionLevel[row][col] = sum;
		}
	}

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += F_functionLevel[i][row] * F_functionLevel[i][col];
			}

			temp[row][col] = sum;
		}
	}

	//printf("%.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n",
	//	F_functionLevel[0][0], F_functionLevel[0][1], F_functionLevel[0][2],
	//	F_functionLevel[1][0], F_functionLevel[1][1], F_functionLevel[1][2],
	//	F_functionLevel[2][0], F_functionLevel[2][1], F_functionLevel[2][2]);

	//1. DIAGONALISE F--------------------------------------------------------------------------------------------------------
	//------------------------------------------------------------------------------------------------------------------------
	double Q[3][3];
	double w[3];
	const int n = 3;
	double sd = 0.0;
	double so = 0.0;                  // Sums of diagonal resp. off-diagonal elements
	double s = 0.0;
	double c = 0.0;
	double t = 0.0;                 // sin(phi), cos(phi), tan(phi) and temporary storage
	double g = 0.0;
	double h = 0.0;
	double z = 0.0;
	double theta = 0.0;          // More temporary storage
	double thresh = 0.0;

	// Initialize Q to the identitity matrix
	for (int i = 0; i < n; i++)
	{
		Q[i][i] = 1.0;
		for (int j = 0; j < i; j++)
			Q[i][j] = Q[j][i] = 0.0;
	}

	// Initialize w to diag(temp)
	for (int i = 0; i < n; i++)
		w[i] = temp[i][i];

	// Calculate SQR(tr(temp))  
	sd = 0.0;
	for (int i = 0; i < n; i++)
		sd += fabs(w[i]);
	sd = sqr(sd);

	// Main iteration loop
	for (int nIter = 0; nIter < 50; nIter++)
	{
		// Test for convergence 
		so = 0.0;
		for (int p = 0; p < n; p++)
		for (int q = p + 1; q < n; q++)
			so += fabs(temp[p][q]);
		if (so == 0.0)
			break;

		if (nIter < 4)
			thresh = 0.2 * so / sqr(n);
		else
			thresh = 0.0;

		// Do sweep
		for (int p = 0; p < n; p++)
		for (int q = p + 1; q < n; q++)
		{
			g = 100.0 * fabs(temp[p][q]);
			if (nIter > 4 && fabs(w[p]) + g == fabs(w[p])
				&& fabs(w[q]) + g == fabs(w[q]))
			{
				temp[p][q] = 0.0;
			}
			else if (fabs(temp[p][q]) > thresh)
			{
				// Calculate Jacobi transformation
				h = w[q] - w[p];
				if (fabs(h) + g == fabs(h))
				{
					t = temp[p][q] / h;
				}
				else
				{
					theta = 0.5 * h / temp[p][q];
					if (theta < 0.0)
						t = -1.0 / (sqrt(1.0 + sqr(theta)) - theta);
					else
						t = 1.0 / (sqrt(1.0 + sqr(theta)) + theta);
				}
				c = 1.0 / sqrt(1.0 + sqr(t));
				s = t * c;
				z = t * temp[p][q];

				// Apply Jacobi transformation
				temp[p][q] = 0.0;
				w[p] -= z;
				w[q] += z;
				for (int r = 0; r < p; r++)
				{
					t = temp[r][p];
					temp[r][p] = c*t - s*temp[r][q];
					temp[r][q] = s*t + c*temp[r][q];
				}
				for (int r = p + 1; r < q; r++)
				{
					t = temp[p][r];
					temp[p][r] = c*t - s*temp[r][q];
					temp[r][q] = s*t + c*temp[r][q];
				}
				for (int r = q + 1; r < n; r++)
				{
					t = temp[p][r];
					temp[p][r] = c*t - s*temp[q][r];
					temp[q][r] = s*t + c*temp[q][r];
				}

				// Update eigenvectors        
				for (int r = 0; r < n; r++)
				{
					t = Q[r][p];
					Q[r][p] = c*t - s*Q[r][q];
					Q[r][q] = s*t + c*Q[r][q];
				}
			}
		}
	}

	//printf("%.4f, %.4f, %.4f", w[0], w[1], w[2]);

	float U[3][3];

	//Correct diagonalised F
	for (int i = 0; i < 3; ++i)
	{
		w[i] = fmaxf(w[i], 0.0f);
	}

	float determinantQ = Q[0][0]
		* (Q[1][1] * Q[2][2] - Q[1][2] * Q[2][1])
		- Q[0][1]
		* (Q[1][0] * Q[2][2] - Q[1][2] * Q[2][0])
		+ Q[0][2]
		* (Q[1][0] * Q[2][1] - Q[1][1] * Q[2][0]);

	//printf("%.4f, ", determinantQ);

	//remove reflection from V if necessary
	if (determinantQ < 0.0f)
	{
		float minElementValue = 1.5e+10f;
		int minElementIdx = 111;
		for (int i = 0; i < 3; ++i)
		{
			if (w[i] < minElementValue)
			{
				minElementValue = w[i];
				minElementIdx = i;
			}
		}
		for (int row = 0; row < 3; ++row)
		{
			Q[row][minElementIdx] *= -1.0f;
		}
	}

	//printf("%.4f, %.4f, %.4f", w[0], w[1], w[2]);

	//determine entries of F
	for (int i = 0; i < 3; ++i)
	{
		w[i] = sqrtf(w[i]);
	}

	//printf("%.4f, %.4f, %.4f", w[0], w[1], w[2]);

	//U = F_orig * V * F.inverse();
	float sum;
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			sum = 0.0f;
			for (int i = 0; i < 3; i++)
			{
				sum += F_functionLevel[row][i] * Q[i][col];
			}

			U[row][col] = sum;
		}
	}

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			sum = 0.0f;
			for (int i = 0; i < 3; i++)
			{
				if (i == col && w[i] != 0.0f)
				{
					sum += U[row][i] * (1.0f / w[i]);
				}
			}

			temp[row][col] = sum;
		}
	}

	for (int row = 0; row < 3; row++)
	{
		for (int col = 0; col < 3; col++)
		{
			U[row][col] = temp[row][col];
		}
	}

	int numEntriesNearZero = 0;
	int idx = 0;
	for (int i = 0; i < 3; ++i)
	{
		if (fabs(w[i]) < 1.0e-4f)
		{
			++numEntriesNearZero;
			idx = i;
		}
	}

	if (numEntriesNearZero > 0)
	{
		if (numEntriesNearZero == 1)
		{
			if (idx == 0)
			{
				//U.col(0) = U.col(1).cross(U.col(2)).normalized();
				U[0][0] = U[1][1] * U[2][2] - U[2][1] * U[1][2];
				U[1][0] = U[2][1] * U[0][2] - U[0][1] * U[2][2];
				U[2][0] = U[0][1] * U[1][2] - U[1][1] * U[0][2];
				U[0][0] /= sqrtf((U[0][0] * U[0][0]) + (U[1][0] * U[1][0]) + (U[2][0] * U[2][0]));
				U[1][0] /= sqrtf((U[0][0] * U[0][0]) + (U[1][0] * U[1][0]) + (U[2][0] * U[2][0]));
				U[2][0] /= sqrtf((U[0][0] * U[0][0]) + (U[1][0] * U[1][0]) + (U[2][0] * U[2][0]));
			}
			else if (idx == 1)
			{
				//U.col(1) = U.col(0).cross(U.col(2)).normalized();
				U[0][1] = U[1][0] * U[2][2] - U[2][0] * U[1][2];
				U[1][1] = U[2][0] * U[0][2] - U[0][0] * U[2][2];
				U[2][1] = U[0][0] * U[1][2] - U[1][0] * U[0][2];
				U[0][1] /= sqrtf((U[0][1] * U[0][1]) + (U[1][1] * U[1][1]) + (U[2][1] * U[2][1]));
				U[1][1] /= sqrtf((U[0][1] * U[0][1]) + (U[1][1] * U[1][1]) + (U[2][1] * U[2][1]));
				U[2][1] /= sqrtf((U[0][1] * U[0][1]) + (U[1][1] * U[1][1]) + (U[2][1] * U[2][1]));
			}
			else
			{
				//U.col(2) = U.col(0).cross(U.col(1)).normalized();
				U[0][2] = U[1][0] * U[2][1] - U[2][0] * U[1][1];
				U[1][2] = U[2][0] * U[0][1] - U[0][0] * U[2][1];
				U[2][2] = U[0][0] * U[1][1] - U[1][0] * U[0][1];
				U[0][2] /= sqrtf((U[0][2] * U[0][2]) + (U[1][2] * U[1][2]) + (U[2][2] * U[2][2]));
				U[1][2] /= sqrtf((U[0][2] * U[0][2]) + (U[1][2] * U[1][2]) + (U[2][2] * U[2][2]));
				U[2][2] /= sqrtf((U[0][2] * U[0][2]) + (U[1][2] * U[1][2]) + (U[2][2] * U[2][2]));
			}
		}
		else
		{
			//set U to identity
			U[0][0] = 1.0f;
			U[1][1] = 1.0f;
			U[2][2] = 1.0f;

			U[0][1] = 0.0f;
			U[0][2] = 0.0f;

			U[1][0] = 0.0f;
			U[1][2] = 0.0f;

			U[2][0] = 0.0f;
			U[2][1] = 0.0f;
		}
	}

	float determinantU = U[0][0]
		* (U[1][1] * U[2][2] - U[1][2] * U[2][1])
		- U[0][1]
		* (U[1][0] * U[2][2] - U[1][2] * U[2][0])
		+ U[0][2]
		* (U[1][0] * U[2][1] - U[1][1] * U[2][0]);

	//remove reflection from U if necessary
	if (determinantU < 0.0f)
	{
		float minElementValue = 1.5e+10f;
		int minElementIdx = 111;
		for (int i = 0; i < 3; ++i)
		{
			if (w[i] < minElementValue)
			{
				minElementValue = w[i];
				minElementIdx = i;
			}
		}

		w[minElementIdx] *= -1.0f;
		for (int row = 0; row < 3; ++row)
		{
			U[row][minElementIdx] *= -1.0f;
		}
	}

	for (int i = 0; i < 3; i++)
	{
		w[i] = fmaxf(w[i], 0.577f);
	}
	//printf("%.4f, %.4f, %.4f", w[0], w[1], w[2]);

	//Store U, V & F
	globalF[IDX * 3 + 0] = w[0];
	globalF[IDX * 3 + 1] = w[1];
	globalF[IDX * 3 + 2] = w[2];

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			globalU[IDX * 9 + row * 3 + col] = U[row][col];
		}
	}

	//printf("%.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n",
	//	U[0][0], U[0][1], U[0][2],
	//	U[1][0], U[1][1], U[1][2],
	//	U[2][0], U[2][1], U[2][2]);

	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			globalV[IDX * 9 + row * 3 + col] = Q[row][col];
		}
	}
	//printf("%.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n %.4f, %.4f, %.4f \n",
	//	Q[0][0], Q[0][1], Q[0][2],
	//	Q[1][0], Q[1][1], Q[1][2],
	//	Q[2][0], Q[2][1], Q[2][2]);

	//printf("%.4f, %.4f, %.4f", w[0], w[1], w[2]);
}


hipError_t projectConstraints(int* device_indices, float* device_positions,
	float* device_inverseMasses, float* device_refShapeMatrixInverses,
	float* device_volumes,
	float* device_F, float* device_U, float* device_V,
	const Parameters& settings);

int CUDA_projectConstraints(int* device_indices, float* device_positions,
	float* device_inverseMasses, float* device_refShapeMatrixInverses,
	float* device_volumes,
	float* device_F, float* device_U, float* device_V,
	const Parameters& settings)
{
	//GPU
	hipError_t cudaStatus = projectConstraints(device_indices, device_positions,
		device_inverseMasses, device_refShapeMatrixInverses, device_volumes, device_F, device_U, device_V, settings);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Critical Error, aborting...");
		return 1;
	}

	return 0;
}

hipError_t cudaErrorWrapper(hipError_t status)
{
	if (status != hipSuccess)
	{
		std::cerr << "ERROR: " << hipGetErrorString(status) << std::endl;
	}

	return status;
}

bool checkCudaErrorStatus(hipError_t status)
{
	if (status != hipSuccess)
	{
		std::cerr << "ERROR: " << hipGetErrorString(status) << std::endl;
		return false;
	}
	return true;
}

void getCudaDeviceProperties(int device)
{
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	std::cout << "Compute Capabilities for " << properties.name << " : " << std::endl;
	std::cout << "Major: " << properties.major << ", Minor: " << properties.minor << std::endl;
	std::cout << "Details: " << std::endl;
	std::cout << "	Num of SM    : " << properties.multiProcessorCount << std::endl;
	std::cout << "	Mem per Block: " << properties.sharedMemPerBlock << std::endl;
	std::cout << "	Mem per SM   : " << properties.sharedMemPerMultiprocessor << std::endl;
}

void queryCUDADevices()
{
	hipError_t deviceStatus;

	int deviceCount = 0;
	deviceStatus = hipGetDeviceCount(&deviceCount);

	std::cout << "Num CUDA Devices Found: " << deviceCount << std::endl;
	deviceStatus = cudaErrorWrapper(hipSetDevice(0));
	checkCudaErrorStatus(deviceStatus);
	getCudaDeviceProperties(0);
}

hipError_t projectConstraints(int* device_indices, float* device_positions,
	float* device_inverseMasses, float* device_refShapeMatrixInverses,
	float* device_volumes,
	float* device_F, float* device_U, float* device_V,
	const Parameters& settings)
{
	float* dev_positions = 0;
	float* dev_inverseMasses = 0;
	int* dev_indices = 0;
	float* dev_refShapeMatrixInverses = 0;
	float* dev_volumes = 0;

	hipError_t deviceStatus;

	//Execute Kernel
	//std::cout << "Executing Kernel..." << settings.numConstraintIterations<< std::endl;
	//std::cout << settings.numBlocks * settings.numThreadsPerBlock << "threads..." << std::endl;
	//std::cout << settings.trueNumberOfConstraints << "true num of constraints..." << std::endl;

	dim3 numBlocks;
	dim3 numThreads;
	numBlocks.x = settings.numBlocks;
	//numBlocks.x = 1;
	numBlocks.y = 1;
	numBlocks.z = 1;

	numThreads.x = settings.numThreadsPerBlock;
	numThreads.y = 1;
	numThreads.z = 1;

	hipEvent_t start;
	hipEvent_t end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);

	for (int it = 0; it < settings.numConstraintIterations; ++it)
	{
		computeDiagonalF << <numBlocks, numThreads >> >(
			device_positions, device_indices,
			device_F, device_U, device_V,
			device_refShapeMatrixInverses,
			settings.trueNumberOfConstraints);

		cudaErrorWrapper(hipDeviceSynchronize());
		//std::cout << "Projection-------------------------------------------------------------------------" << std::endl;

		solveFEMConstraint << <numBlocks, numThreads >> >(
			device_positions, device_indices, device_inverseMasses,
			device_volumes, device_refShapeMatrixInverses,
			settings.lambda, settings.mu, settings.trueNumberOfConstraints,
			settings.numParticles,
			device_F, device_U, device_V);

		cudaErrorWrapper(hipDeviceSynchronize());
		//std::cout << "Projection done-------------------------------------------------------------------------" << std::endl;
	}
	cudaErrorWrapper(hipDeviceSynchronize());

	hipEventRecord(end);
	hipEventSynchronize(end);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, end);
	std::cout << "Execution Time: " << milliseconds / 1000.0 << "s." << std::endl;

	deviceStatus = hipGetLastError();
	checkCudaErrorStatus(deviceStatus);

	return deviceStatus;
}

bool CUDA_allocateBuffers(int** device_indices, float** device_positions,
	float** device_inverseMasses, float** device_refShapeMatrixInverses,
	float** device_volumes,
	float** device_F, float** device_U, float** device_V,
	std::vector<int>& indices,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& F,
	std::vector<float>& U,
	std::vector<float>& V)
{
	std::cout << "Allocating CUDA Buffers" << std::endl;
	hipError_t deviceStatus;

	deviceStatus = hipSetDevice(0);
	deviceStatus = hipGetLastError();
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}

	deviceStatus = hipMalloc((void**)device_indices, indices.size() * sizeof(int));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_positions, positions.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_inverseMasses, inverseMasses.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_refShapeMatrixInverses, refShapeMatrixInverses.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_volumes, volumes.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_F, F.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_U, U.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMalloc((void**)device_V, V.size() * sizeof(float));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}

	std::cout << "Copying Memory to CUDA device..." << std::endl;
	deviceStatus = hipMemcpy(*device_indices, &indices[0], indices.size() * sizeof(int), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_positions, &positions[0], positions.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_inverseMasses, &inverseMasses[0], inverseMasses.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_refShapeMatrixInverses, &refShapeMatrixInverses[0], refShapeMatrixInverses.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_volumes, &volumes[0], volumes.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_F, &F[0], F.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_U, &U[0], U.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	deviceStatus = hipMemcpy(*device_V, &V[0], V.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}


	//Print some statistics
	double totalNumbytes = indices.size() * sizeof(int)
		+positions.size() * sizeof(float)
		+volumes.size() * sizeof(float)
		+inverseMasses.size() * sizeof(float)
		+refShapeMatrixInverses.size() * sizeof(float)
		+F.size() * sizeof(float)
		+U.size() * sizeof(float)
		+V.size() * sizeof(float);

	std::cout << "Memory Usage: " << std::endl;
	std::cout << "	Indices (int32)             : " << indices.size() * sizeof(int) << " bytes" << std::endl;
	std::cout << "	Positions (float)           : " << positions.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	Volumes (float)             : " << volumes.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	Masses (float)              : " << inverseMasses.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	Ref. Shape Matrices: (float): " << refShapeMatrixInverses.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	F...........................: " << F.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	U...........................: " << U.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	V...........................: " << V.size() * sizeof(float) << " bytes" << std::endl;
	std::cout << "	--------------------------------------------------------" << std::endl;
	std::cout << "	Total                       : " << totalNumbytes << " bytes (" << totalNumbytes / 1000.0 << " kb)" << std::endl;
}

bool CUDA_destroyBuffers(int* device_indices, float* device_positions,
	float* device_inverseMasses, float* device_refShapeMatrixInverses,
	float* device_volumes,
	float* device_F, float* device_U, float* device_V)
{
	hipError_t deviceStatus;

	hipFree(device_positions);
	hipFree(device_inverseMasses);
	hipFree(device_indices);
	hipFree(device_refShapeMatrixInverses);
	hipFree(device_volumes);
	hipFree(device_F);
	hipFree(device_U);
	hipFree(device_V);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	deviceStatus = hipDeviceReset();
	if (deviceStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	deviceStatus = hipGetLastError();
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool CUDA_updateBuffers(float* device_positions, std::vector<float>& positions)
{
	hipError_t deviceStatus;

	deviceStatus = hipMemcpy(device_positions, &positions[0], positions.size() * sizeof(float), hipMemcpyHostToDevice);
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}

	return true;
}

bool CUDA_getBuffers(float* device_positions, std::vector<float>& positions)
{
	hipError_t deviceStatus;

	deviceStatus = cudaErrorWrapper(hipMemcpy(&positions[0], device_positions, positions.size() * sizeof(float), hipMemcpyDeviceToHost));
	if (!checkCudaErrorStatus(deviceStatus))
	{
		return false;
	}

	return true;
}