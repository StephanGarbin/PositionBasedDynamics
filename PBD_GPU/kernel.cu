
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <vector>
#include <iostream>

const int NUM_THREADS_PER_BLOCK_SINGLE = 8;
const int NUM_THREADS_PER_BLOCK = NUM_THREADS_PER_BLOCK_SINGLE * NUM_THREADS_PER_BLOCK_SINGLE;

__shared__ float F[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float FTransposeF[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float FInverseTranspose[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float FirstPiolaKirchoffTensor[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float Gradient[NUM_THREADS_PER_BLOCK][3][4];
__shared__ int LocalIndices[NUM_THREADS_PER_BLOCK][4];


__device__ float sqr(float x)
{
	return x * x;
}

__device__ float traceFTransposeF(int idx)
{
	return FTransposeF[idx][0][0] + FTransposeF[idx][1][1] + FTransposeF[idx][2][2];
}

__device__ float determinantFTransposeF(int idx)
{
	return FTransposeF[idx][0][0]
		* (FTransposeF[idx][1][1] * FTransposeF[idx][2][2] - FTransposeF[idx][1][2] * FTransposeF[idx][2][1])
		- FTransposeF[idx][0][1]
		* (FTransposeF[idx][1][0] * FTransposeF[idx][2][2] - FTransposeF[idx][1][2] * FTransposeF[idx][2][0])
		+ FTransposeF[idx][0][2]
		* (FTransposeF[idx][1][0] * FTransposeF[idx][2][1] - FTransposeF[idx][1][1] * FTransposeF[idx][2][0]);
}

__device__ float determinantFInverseTranspose(int idx)
{
	return FInverseTranspose[idx][0][0]
		* (FInverseTranspose[idx][1][1] * FInverseTranspose[idx][2][2] - FInverseTranspose[idx][1][2] * FInverseTranspose[idx][2][1])
		- FInverseTranspose[idx][0][1]
		* (FInverseTranspose[idx][1][0] * FInverseTranspose[idx][2][2] - FInverseTranspose[idx][1][2] * FInverseTranspose[idx][2][0])
		+ FInverseTranspose[idx][0][2]
		* (FInverseTranspose[idx][1][0] * FInverseTranspose[idx][2][1] - FInverseTranspose[idx][1][1] * FInverseTranspose[idx][2][0]);
}

__device__ void calculateF(int idx, float* positions, float* refShapeMatrixInverse)
{
	//1. Calculate Deformed Shape Matrix
	FirstPiolaKirchoffTensor[idx][0][0] = positions[LocalIndices[idx][0] * 3 + 0] - positions[LocalIndices[idx][3] * 3 + 0];
	FirstPiolaKirchoffTensor[idx][1][0] = positions[LocalIndices[idx][0] * 3 + 1] - positions[LocalIndices[idx][3] * 3 + 1];
	FirstPiolaKirchoffTensor[idx][2][0] = positions[LocalIndices[idx][0] * 3 + 2] - positions[LocalIndices[idx][3] * 3 + 2];

	FirstPiolaKirchoffTensor[idx][0][1] = positions[LocalIndices[idx][1] * 3 + 0] - positions[LocalIndices[idx][3] * 3 + 0];
	FirstPiolaKirchoffTensor[idx][1][1] = positions[LocalIndices[idx][1] * 3 + 1] - positions[LocalIndices[idx][3] * 3 + 1];
	FirstPiolaKirchoffTensor[idx][2][1] = positions[LocalIndices[idx][1] * 3 + 2] - positions[LocalIndices[idx][3] * 3 + 2];

	FirstPiolaKirchoffTensor[idx][0][2] = positions[LocalIndices[idx][2] * 3 + 0] - positions[LocalIndices[idx][3] * 3 + 0];
	FirstPiolaKirchoffTensor[idx][1][2] = positions[LocalIndices[idx][2] * 3 + 1] - positions[LocalIndices[idx][3] * 3 + 1];
	FirstPiolaKirchoffTensor[idx][2][2] = positions[LocalIndices[idx][2] * 3 + 2] - positions[LocalIndices[idx][3] * 3 + 2];

	//2. Multiply 
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += FirstPiolaKirchoffTensor[idx][row][i] * refShapeMatrixInverse[idx * 3 * 3 + row * 3 + col];
			}

			F[idx][row][col] = sum;
		}
	}

}

__device__ void calculateFirstPiolaKirchoffTensor_NEO_HOOKEAN(int idx, float mu, float lambda, float I3)
{
	//1. Copy over F
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] = F[idx][row][col];
		}
	}

	//2. Multiply with mu
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] *= mu;
		}
	}

	//3. Subtract mu times FInverseTranspose
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] -= mu * FInverseTranspose[idx][row][col];
		}
	}

	//4. Add (ambda * logI3) / 2.0 * FInverseTranspose
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] += ((lambda * log(I3)) / 2.0f) * FInverseTranspose[idx][row][col];
		}
	}
}

__device__ float calculateStrainEnergy_NEO_HOOKEAN(float volume, float lambda, float mu, float I1, float I3)
{
	return volume * (0.5f * mu * (I1 - log(I3) - 3.0f) + (lambda / 8.0f) * (log(I3) * log(I3)));
}

__device__ void calculateStrainEnergyGradient_NEO_HOOKEAN(int idx, float volume, float* refShapeMatrixInverse)
{
	//1. Copy refShapeMatrixInverse from global memory
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			Gradient[idx][row][col] = refShapeMatrixInverse[idx * 3 + row * 3 + col];
		}
	}

	//2. Multiply by volume
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			Gradient[idx][row][col] *= volume;
		}
	}

	//3. Multiply with First Piola-Kirchoff Stress tensor
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += Gradient[idx][row][i] * FirstPiolaKirchoffTensor[idx][i][row];
			}

			FTransposeF[idx][row][col] = sum;
		}
	}

	//4. Copy back
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			Gradient[idx][row][col] = FTransposeF[idx][row][col];
		}
	}


	//4. Calculate last column
	for (int row = 0; row < 3; ++row)
	{
		float sum = 0.0f;
		for (int col = 0; col < 3; ++col)
		{
			sum += Gradient[idx][row][col];
		}
		Gradient[idx][row][3] = -sum;
	}
}

__device__ void calculateFTransposeF(int idx)
{
	//1. Copy over F (1&2 could be combined in future!)
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FTransposeF[idx][row][col] = F[idx][row][col];
		}
	}

	//2. Transpose F
	float temp;
	temp = FTransposeF[idx][0][1];
	FTransposeF[idx][0][1] = FTransposeF[idx][1][0];
	FTransposeF[idx][1][0] = temp;

	temp = FTransposeF[idx][0][2];
	FTransposeF[idx][0][2] = FTransposeF[idx][2][0];
	FTransposeF[idx][2][0] = temp;

	temp = FTransposeF[idx][1][2];
	FTransposeF[idx][1][2] = FTransposeF[idx][2][1];
	FTransposeF[idx][2][1] = temp;

	//3. Multiply with F
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += FTransposeF[idx][row][i] * F[idx][i][row];
			}

			FirstPiolaKirchoffTensor[idx][row][col] = sum;
		}
	}

	//Copy back
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FTransposeF[idx][row][col] = FirstPiolaKirchoffTensor[idx][row][col];
		}
	}
}

__device__ void calculateFInverseTranspose(int idx)
{
	//1. Calculate cofactors
	FInverseTranspose[idx][0][0] = F[idx][1][1] * F[idx][2][2] - F[idx][2][1] * F[idx][1][2];
	FInverseTranspose[idx][0][1] = F[idx][1][0] * F[idx][2][2] - F[idx][2][0] * F[idx][1][2];
	FInverseTranspose[idx][0][2] = F[idx][1][0] * F[idx][2][1] - F[idx][2][0] * F[idx][1][1];

	FInverseTranspose[idx][1][0] = F[idx][0][1] * F[idx][2][2] - F[idx][2][1] * F[idx][0][2];
	FInverseTranspose[idx][1][1] = F[idx][0][0] * F[idx][2][2] - F[idx][2][0] * F[idx][0][2];
	FInverseTranspose[idx][1][2] = F[idx][0][0] * F[idx][2][1] - F[idx][2][0] * F[idx][0][1];

	FInverseTranspose[idx][2][0] = F[idx][1][1] * F[idx][1][2] - F[idx][2][1] * F[idx][0][2];
	FInverseTranspose[idx][2][1] = F[idx][1][0] * F[idx][1][2] - F[idx][2][0] * F[idx][0][2];
	FInverseTranspose[idx][2][2] = F[idx][1][0] * F[idx][1][1] - F[idx][2][0] * F[idx][0][1];

	//2. Transpose
	float temp;

	temp = FInverseTranspose[idx][0][1];
	FInverseTranspose[idx][0][1] = FInverseTranspose[idx][1][0];
	FInverseTranspose[idx][1][0] = temp;

	temp = FInverseTranspose[idx][0][2];
	FInverseTranspose[idx][0][2] = FInverseTranspose[idx][2][0];
	FInverseTranspose[idx][2][0] = temp;

	temp = FInverseTranspose[idx][1][2];
	FInverseTranspose[idx][1][2] = FInverseTranspose[idx][2][1];
	FInverseTranspose[idx][2][1] = temp;

	//3. Calculate the determinant
	float determinant = determinantFInverseTranspose(idx);

	//4. Multiply
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FInverseTranspose[idx][row][col] /= determinant;
		}
	}
}

__device__ float squaredNormGradient(int idx, int particleIdx)
{
	return sqrtf(sqr(Gradient[idx][0][particleIdx])
		+ sqr(Gradient[idx][1][particleIdx])
		+ sqr(Gradient[idx][2][particleIdx]));
}

__device__ float calculateLagrangeMultiplierDenominator(int idx, float* inverseMass)
{
	float denominator = 0.0f;
	for (int i = 0; i < 4; ++i)
	{
		denominator += inverseMass[idx + i] * squaredNormGradient(idx, i);
	}

	return denominator;
}

__device__ void updatePositions(int idx, float lagrangeMultiplier, float* positions, float* inverseMass)
{
	for (int i = 0; i < 4; ++i)
	{
		/*atomicAdd(&positions[LocalIndices[idx][i] * 3 + 0], inverseMass[idx] * lagrangeMultiplier * Gradient[idx][0][i]);
		atomicAdd(&positions[LocalIndices[idx][i] * 3 + 1], inverseMass[idx] * lagrangeMultiplier * Gradient[idx][1][i]);
		atomicAdd(&positions[LocalIndices[idx][i] * 3 + 2], inverseMass[idx] * lagrangeMultiplier * Gradient[idx][2][i]);*/
	}
}

__device__ void getIndices(int idx, int* indices)
{
	for (int i = 0; i < 4; ++i)
	{
		LocalIndices[idx][i] = indices[idx * 4 + i];
	}
}

__global__ void solveFEMConstraint(float* positions, int* indices, float* inverseMass, float* volume, float* refShapeMatrixInverse,
	float lambda, float mu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//1. Calculate Deformation Gradient F
	calculateF(idx, positions, refShapeMatrixInverse);

	//2. Compute Cauchy Tensors
	calculateFInverseTranspose(idx);
	calculateFTransposeF(idx);

	//3. Compute Invariants
	float I1 = traceFTransposeF(idx);
	float I3 = determinantFTransposeF(idx);

	//4. Calculate First Piola-Kirchoff Stress Tensor
	calculateFirstPiolaKirchoffTensor_NEO_HOOKEAN(idx, mu, lambda, I3);

	//5. Calculate StrainEnergy
	float strainEnergy = calculateStrainEnergy_NEO_HOOKEAN(volume[idx], lambda, mu, I1, I3);

	//6. Calculate Strain Energy Gradient
	calculateStrainEnergyGradient_NEO_HOOKEAN(idx, volume[idx], refShapeMatrixInverse);

	//7. Calculate Lagrange Multiplier
	float lagrangeMultiplier = - (strainEnergy / calculateLagrangeMultiplierDenominator(idx, inverseMass));

	//8. Update Positions
	updatePositions(idx, lagrangeMultiplier, positions, inverseMass);
}



hipError_t projectConstraints(std::vector<int>& indices,
	std::vector<float>& originalPositions,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& positions_result,
	float lambda, float mu);

void projectConstraintsHOST(std::vector<int>& indices,
	std::vector<float>& originalPositions,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& positions_result,
	float lambda, float mu);


void setUpSystem(std::vector<int>& indices, std::vector<float>& originalPositions,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	float gravity, float deltaT)
{
	originalPositions.push_back(0.0f); originalPositions.push_back(0.0f); originalPositions.push_back(0.0f);
	originalPositions.push_back(-0.946f); originalPositions.push_back(0.0f); originalPositions.push_back(-1.114f);
	originalPositions.push_back(0.689f); originalPositions.push_back(0.515f); originalPositions.push_back(-1.114f);
	originalPositions.push_back(0.689f); originalPositions.push_back(-0.757f); originalPositions.push_back(-1.114f);
	originalPositions.push_back(0.0f); originalPositions.push_back(0.0f); originalPositions.push_back(-2.576f);

	indices.push_back(3); indices.push_back(0); indices.push_back(2); indices.push_back(1);
	indices.push_back(3); indices.push_back(4); indices.push_back(1); indices.push_back(2);

	for (int i = 0; i < 5; ++i)
	{
		inverseMasses.push_back(1.0f);
	}
	inverseMasses[0] = 0.0f;

	for (int i = 0; i < originalPositions.size(); ++i)
	{
		positions.push_back(originalPositions[i]);
	}

	//apply one time step of deformations
	for (int i = 0; i < 5; ++i)
	{
		positions[i * 3 + 1] += inverseMasses[i] * gravity * deltaT;
	}

	//FROM MATLAB
	volumes.push_back(0.38613f);
	volumes.push_back(0.50676f);


}


int main()
{
	std::vector<int> indices;
	std::vector<float> originalPositions;
	std::vector<float> positions;
	std::vector<float> inverseMasses;
	std::vector<float> refShapeMatrixInverses;
	std::vector<float> volumes;


	float deltaT = 0.005f;
	float gravity = -9.8f;

	float lambda = 0.769231f;
	float mu = 1.15385f;

	setUpSystem(indices, originalPositions, positions, inverseMasses, refShapeMatrixInverses, volumes, gravity, deltaT);

	std::vector<float> positionsResultDevice(positions.size());
	std::vector<float> positionsResultHost(positions.size());

	//CPU
	projectConstraintsHOST(indices, originalPositions, positions,
		inverseMasses, refShapeMatrixInverses, volumes, positionsResultHost, lambda, mu);
	
	//GPU
	hipError_t cudaStatus = projectConstraints(indices, originalPositions, positions,
		inverseMasses, refShapeMatrixInverses, volumes, positionsResultDevice, lambda, mu);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Critical Error, aborting...");
		return 1;
	}

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//Print Some Results

	std::cout << "AFTER PROJECION HOST: " << std::endl;
	for (int row = 0; row < 5; +row)
	{
		for (int col = 0; col < 3; ++col)
		{
			std::cout << positionsResultHost[row * 3 + col] << ", ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl << std::endl;

	std::cout << "AFTER PROJECION DEVICE: " << std::endl;
	for (int row = 0; row < 5; +row)
	{
		for (int col = 0; col < 3; ++col)
		{
			std::cout << positionsResultDevice[row * 3 + col] << ", ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl << std::endl;

	return 0;
}

hipError_t cudaErrorWrapper(hipError_t status)
{
	if (status != hipSuccess) {
		fprintf(stderr, "Critical Error occured!");
		std::cout << "ERROR Details: " << hipGetErrorString(status) << std::endl;
	}

	return status;
}

void getCudaDeviceProperties(int device)
{
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	std::cout << "Compute Capabilities for " << properties.name << " : " << std::endl;
	std::cout << "Major: " << properties.major << ", Minor: " << properties.minor << std::endl;
	std::cout << "Details: " << std::endl;
	std::cout << "	Num of SM    : " << properties.multiProcessorCount << std::endl;
	std::cout << "	Mem per Block: " << properties.sharedMemPerBlock << std::endl;
	std::cout << "	Mem per SM   : " << properties.sharedMemPerMultiprocessor << std::endl;
}

hipError_t projectConstraints(std::vector<int>& indices, std::vector<float>& originalPositions,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& positions_result,
	float lambda, float mu)
{
	float* dev_positions;
	float* dev_inverseMasses;
	int* dev_indices;

	float* dev_refShapeMatrixInverses;
	float* dev_volumes;

	hipError_t deviceStatus;

	//Allocate memory
	int deviceCount = 0;
	deviceStatus == hipGetDeviceCount(&deviceCount);

	std::cout << "Num CUDA Devices Found: " << deviceCount << std::endl;
	deviceStatus = cudaErrorWrapper(hipSetDevice(0));
	getCudaDeviceProperties(0);

	std::cout << "Calling CUDA Malloc..." << std::endl;

	deviceStatus = cudaErrorWrapper(hipMalloc((void**)&dev_indices, indices.size() * sizeof(int)));
	deviceStatus = cudaErrorWrapper(hipMalloc((void**)&dev_positions, positions.size() * sizeof(float)));
	deviceStatus = cudaErrorWrapper(hipMalloc((void**)&dev_inverseMasses, inverseMasses.size() * sizeof(float)));
	deviceStatus = cudaErrorWrapper(hipMalloc((void**)&dev_refShapeMatrixInverses, refShapeMatrixInverses.size() * sizeof(float)));
	deviceStatus = cudaErrorWrapper(hipMalloc((void**)&dev_volumes, volumes.size() * sizeof(float)));

	//Cpy memory
	deviceStatus = cudaErrorWrapper(hipMemcpy(dev_indices, &indices[0], indices.size() * sizeof(int), hipMemcpyHostToDevice));
	deviceStatus = cudaErrorWrapper(hipMemcpy(dev_positions, &positions[0], positions.size() * sizeof(float), hipMemcpyHostToDevice));
	deviceStatus = cudaErrorWrapper(hipMemcpy(dev_inverseMasses, &inverseMasses[0], inverseMasses.size() * sizeof(float), hipMemcpyHostToDevice));
	deviceStatus = cudaErrorWrapper(hipMemcpy(dev_refShapeMatrixInverses, &refShapeMatrixInverses[0], refShapeMatrixInverses.size() * sizeof(float), hipMemcpyHostToDevice));
	deviceStatus = cudaErrorWrapper(hipMemcpy(dev_volumes, &volumes[0], volumes.size() * sizeof(float), hipMemcpyHostToDevice));

	//Execute Kernel
	//solveFEMConstraint<<<1, 2>>>(dev_positions, dev_indices, dev_inverseMasses, dev_volumes, dev_refShapeMatrixInverses, lambda, mu);

	hipDeviceSynchronize();

	//Cpy memory back
	positions_result.resize(positions.size());
	deviceStatus = cudaErrorWrapper(hipMemcpy(&positions_result[0], dev_positions, positions_result.size() * sizeof(float), hipMemcpyDeviceToHost));

	//Free memory
	hipFree(dev_positions);
	hipFree(dev_inverseMasses);
	hipFree(dev_indices);
	hipFree(dev_refShapeMatrixInverses);
	hipFree(dev_volumes);
	return deviceStatus;
}


void projectConstraintsHOST(std::vector<int>& indices,
	std::vector<float>& originalPositions,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& positions_result,
	float lambda, float mu)
{
	positions_result.push_back(0.000000000000000f);
	positions_result.push_back(-4.900000000000000f);
	positions_result.push_back(0.000000000000000f);
	positions_result.push_back(-0.946000000000000f);
	positions_result.push_back(-4.900000000000000f);
	positions_result.push_back(-1.114000000000000f);
	positions_result.push_back(0.689000000000000f);
	positions_result.push_back(-4.385000000000001f);
	positions_result.push_back(-1.114000000000000f);
	positions_result.push_back(0.689000000000000f);
	positions_result.push_back(-5.657000000000000f);
	positions_result.push_back(-1.114000000000000f);
	positions_result.push_back(0.000000000000000f);
	positions_result.push_back(-4.900000000000000f);
	positions_result.push_back(-2.576000000000000f);
}