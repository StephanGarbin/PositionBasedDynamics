#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>

#include "CUDA_WRAPPER.h"

const int NUM_THREADS_PER_BLOCK_SINGLE = 8;
const int NUM_THREADS_PER_BLOCK = NUM_THREADS_PER_BLOCK_SINGLE * NUM_THREADS_PER_BLOCK_SINGLE;

__shared__ float F[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float FTransposeF[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float FInverseTranspose[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float FirstPiolaKirchoffTensor[NUM_THREADS_PER_BLOCK][3][3];
__shared__ float Gradient[NUM_THREADS_PER_BLOCK][3][4];
__shared__ int LocalIndices[NUM_THREADS_PER_BLOCK][4];
__shared__ float LocalMasses[NUM_THREADS_PER_BLOCK][4];


__device__ float sqr(float x)
{
	return x * x;
}

__device__ float traceFTransposeF(int idx)
{
	return FTransposeF[idx][0][0] + FTransposeF[idx][1][1] + FTransposeF[idx][2][2];
}

__device__ float determinantFTransposeF(int idx)
{
	return FTransposeF[idx][0][0]
		* (FTransposeF[idx][1][1] * FTransposeF[idx][2][2] - FTransposeF[idx][1][2] * FTransposeF[idx][2][1])
		- FTransposeF[idx][0][1]
		* (FTransposeF[idx][1][0] * FTransposeF[idx][2][2] - FTransposeF[idx][1][2] * FTransposeF[idx][2][0])
		+ FTransposeF[idx][0][2]
		* (FTransposeF[idx][1][0] * FTransposeF[idx][2][1] - FTransposeF[idx][1][1] * FTransposeF[idx][2][0]);
}

__device__ float determinantF(int idx)
{
	return F[idx][0][0]
		* (F[idx][1][1] * F[idx][2][2] - F[idx][1][2] * F[idx][2][1])
		- F[idx][0][1]
		* (F[idx][1][0] * F[idx][2][2] - F[idx][1][2] * F[idx][2][0])
		+ F[idx][0][2]
		* (F[idx][1][0] * F[idx][2][1] - F[idx][1][1] * F[idx][2][0]);
}

__device__ void calculateF(int idx, float* positions, float* refShapeMatrixInverse)
{
	//1. Calculate Deformed Shape Matrix
	FirstPiolaKirchoffTensor[idx][0][0] = positions[LocalIndices[idx][0] * 3 + 0] - positions[LocalIndices[idx][3] * 3 + 0];
	FirstPiolaKirchoffTensor[idx][1][0] = positions[LocalIndices[idx][0] * 3 + 1] - positions[LocalIndices[idx][3] * 3 + 1];
	FirstPiolaKirchoffTensor[idx][2][0] = positions[LocalIndices[idx][0] * 3 + 2] - positions[LocalIndices[idx][3] * 3 + 2];

	FirstPiolaKirchoffTensor[idx][0][1] = positions[LocalIndices[idx][1] * 3 + 0] - positions[LocalIndices[idx][3] * 3 + 0];
	FirstPiolaKirchoffTensor[idx][1][1] = positions[LocalIndices[idx][1] * 3 + 1] - positions[LocalIndices[idx][3] * 3 + 1];
	FirstPiolaKirchoffTensor[idx][2][1] = positions[LocalIndices[idx][1] * 3 + 2] - positions[LocalIndices[idx][3] * 3 + 2];

	FirstPiolaKirchoffTensor[idx][0][2] = positions[LocalIndices[idx][2] * 3 + 0] - positions[LocalIndices[idx][3] * 3 + 0];
	FirstPiolaKirchoffTensor[idx][1][2] = positions[LocalIndices[idx][2] * 3 + 1] - positions[LocalIndices[idx][3] * 3 + 1];
	FirstPiolaKirchoffTensor[idx][2][2] = positions[LocalIndices[idx][2] * 3 + 2] - positions[LocalIndices[idx][3] * 3 + 2];

	//printf("Local Indices: \n");
	//for (int i = 0; i < 4; ++i)
	//{
	//	printf("%d, ", LocalIndices[idx][i]);
	//}
	//printf("\n");
	//
	//printf("Particles: \n");
	//for (int i = 0; i < 4; ++i)
	//{
	//	printf("%4.4f ,", positions[LocalIndices[idx][i] * 3 + 0]);
	//	printf("%4.4f ,", positions[LocalIndices[idx][i] * 3 + 1]);
	//	printf("%4.4f \n", positions[LocalIndices[idx][i] * 3 + 2]);
	//}
	//printf("Particles END \n");
	//printf("\n");
	//printf("Ref Shape Matrix: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		printf("%4.4f,", refShapeMatrixInverse[idx * 3 * 3 + row * 3 + col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");

	//2. Multiply 
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += FirstPiolaKirchoffTensor[idx][row][i] * refShapeMatrixInverse[idx * 3 * 3 + i * 3 + col];
			}

			F[idx][row][col] = sum;
		}
	}
}

__device__ void calculateFirstPiolaKirchoffTensor_NEO_HOOKEAN(int idx, float mu, float lambda, float I3)
{
	//1. Copy over F multiplied with mu
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] = F[idx][row][col] * mu;
		}
	}

	//3. Subtract mu times FInverseTranspose
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] -= FInverseTranspose[idx][row][col] * mu;
		}
	}

	//4. Add (lambda * logI3) / 2.0 * FInverseTranspose
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FirstPiolaKirchoffTensor[idx][row][col] += FInverseTranspose[idx][row][col] * ((lambda * log(I3)) / 2.0f);
		}
	}
}

__device__ float calculateStrainEnergy_NEO_HOOKEAN(float volume, float lambda, float mu, float I1, float I3)
{
	return volume * (0.5f * mu * (I1 - log(I3) - 3.0f) + (lambda / 8.0f) * (log(I3) * log(I3)));
}

__device__ void calculateStrainEnergyGradient_NEO_HOOKEAN(int idx, float volume, float* refShapeMatrixInverse)
{
	//1. Copy refShapeMatrixInverse from global memory
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			Gradient[idx][row][col] = refShapeMatrixInverse[idx * 3 + row * 3 + col];
		}
	}

	//2. Multiply by volume
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			Gradient[idx][row][col] *= volume;
		}
	}

	//3. Multiply with First Piola-Kirchoff Stress tensor
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += Gradient[idx][row][i] * FirstPiolaKirchoffTensor[idx][i][col];
			}

			FTransposeF[idx][col][row] = sum;
		}
	}

	//4. Copy back
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			Gradient[idx][row][col] = FTransposeF[idx][row][col];
		}
	}


	//4. Calculate last column
	for (int row = 0; row < 3; ++row)
	{
		float sum = 0.0f;
		for (int col = 0; col < 3; ++col)
		{
			sum += Gradient[idx][row][col];
		}
		Gradient[idx][row][3] = -sum;
	}
}

__device__ void calculateFTransposeF(int idx)
{
	//Combine all into one loop in future!

	//1. Copy over F
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FTransposeF[idx][row][col] = F[idx][row][col];
		}
	}

	//2. Transpose F (Subsume into multiplication later!)
	float temp;
	temp = FTransposeF[idx][0][1];
	FTransposeF[idx][0][1] = FTransposeF[idx][1][0];
	FTransposeF[idx][1][0] = temp;

	temp = FTransposeF[idx][0][2];
	FTransposeF[idx][0][2] = FTransposeF[idx][2][0];
	FTransposeF[idx][2][0] = temp;

	temp = FTransposeF[idx][1][2];
	FTransposeF[idx][1][2] = FTransposeF[idx][2][1];
	FTransposeF[idx][2][1] = temp;


	//printf("FTranspose: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		printf("%4.8f,", FTransposeF[idx][row][col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");

	//3. Multiply with F
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			float sum = 0.0f;

			for (int i = 0; i < 3; ++i)
			{
				sum += FTransposeF[idx][row][i] * F[idx][i][col];
			}

			FirstPiolaKirchoffTensor[idx][row][col] = sum;
		}
	}

	//Copy back
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FTransposeF[idx][row][col] = FirstPiolaKirchoffTensor[idx][row][col];
		}
	}
}

__device__ void calculateFInverseTranspose(int idx)
{
	//1. Calculate cofactors
	FInverseTranspose[idx][0][0] = F[idx][1][1] * F[idx][2][2] - F[idx][2][1] * F[idx][1][2];
	FInverseTranspose[idx][0][1] = -(F[idx][1][0] * F[idx][2][2] - F[idx][2][0] * F[idx][1][2]);
	FInverseTranspose[idx][0][2] = F[idx][1][0] * F[idx][2][1] - F[idx][2][0] * F[idx][1][1];

	FInverseTranspose[idx][1][0] = -(F[idx][0][1] * F[idx][2][2] - F[idx][2][1] * F[idx][0][2]);
	FInverseTranspose[idx][1][1] = F[idx][0][0] * F[idx][2][2] - F[idx][2][0] * F[idx][0][2];
	FInverseTranspose[idx][1][2] = -(F[idx][0][0] * F[idx][2][1] - F[idx][2][0] * F[idx][0][1]);

	FInverseTranspose[idx][2][0] = F[idx][0][1] * F[idx][1][2] - F[idx][1][1] * F[idx][0][2];
	FInverseTranspose[idx][2][1] = -(F[idx][0][0] * F[idx][1][2] - F[idx][1][0] * F[idx][0][2]);
	FInverseTranspose[idx][2][2] = F[idx][0][0] * F[idx][1][1] - F[idx][1][0] * F[idx][0][1];

	//2. Transpose (Alread in Co-factor calculation)
	//float temp;

	//temp = FInverseTranspose[idx][0][1];
	//FInverseTranspose[idx][0][1] = FInverseTranspose[idx][1][0];
	//FInverseTranspose[idx][1][0] = temp;

	//temp = FInverseTranspose[idx][0][2];
	//FInverseTranspose[idx][0][2] = FInverseTranspose[idx][2][0];
	//FInverseTranspose[idx][2][0] = temp;

	//temp = FInverseTranspose[idx][1][2];
	//FInverseTranspose[idx][1][2] = FInverseTranspose[idx][2][1];
	//FInverseTranspose[idx][2][1] = temp;

	//3. Calculate the determinant
	float determinant = determinantF(idx);
	//printf("Determinant of F: %4.8f \n", determinant);

	//4. Multiply
	for (int row = 0; row < 3; ++row)
	{
		for (int col = 0; col < 3; ++col)
		{
			FInverseTranspose[idx][row][col] /= determinant;
		}
	}
}

__device__ float squaredNormGradient(int idx, int particleIdx)
{
	return sqrtf(sqr(Gradient[idx][0][particleIdx])
		+ sqr(Gradient[idx][1][particleIdx])
		+ sqr(Gradient[idx][2][particleIdx]));
}

__device__ float calculateLagrangeMultiplierDenominator(int idx, float* inverseMass)
{
	float denominator = 0.0f;
	for (int i = 0; i < 4; ++i)
	{
		denominator += LocalMasses[idx][i] * squaredNormGradient(idx, i);
		//printf("Denominator Component: %4.8f \n", inverseMass[LocalIndices[idx][i]] * squaredNormGradient(idx, i));
	}
	//printf("Denominator: %4.8f \n", denominator);
	return denominator;
}

__device__ void updatePositions(int idx, float lagrangeMultiplier, float* positions, float* inverseMass)
{
	for (int i = 0; i < 4; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			atomicAdd(&positions[LocalIndices[idx][i] * 3 + j], LocalMasses[idx][i] * lagrangeMultiplier * Gradient[idx][j][i]);
			//printf("Position Update %4.8f \n", LocalMasses[idx][i] * lagrangeMultiplier * Gradient[idx][j][i]);
		}
		printf("\n");
	}
}

__device__ void getIndices(int idx, int* indices)
{
	for (int i = 0; i < 4; ++i)
	{
		LocalIndices[idx][i] = indices[idx * 4 + i];
	}
}

__device__ void getMasses(int idx, float* masses)
{
	for (int i = 0; i < 4; ++i)
	{
		LocalMasses[idx][i] = masses[LocalIndices[idx][i]];
	}
}

__global__ void solveFEMConstraint(float* positions, int* indices, float* inverseMass, float* volume, float* refShapeMatrixInverse,
	float lambda, float mu, int trueNumConstraints)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > 10)
	{
		return;
	}

	getIndices(idx, indices);
	getMasses(idx, inverseMass);

	//1. Calculate Deformation Gradient F
	calculateF(idx, positions, refShapeMatrixInverse);

	//printf("F: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		printf("%4.8f,", F[idx][row][col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");

	//2. Compute Cauchy Tensors
	calculateFInverseTranspose(idx);

	//printf("FInverseTranspose: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		printf("%4.8f,", FInverseTranspose[idx][row][col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");


	calculateFTransposeF(idx);

	//printf("FTransposeF: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		printf("%4.8f,", FTransposeF[idx][row][col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");

	//3. Compute Invariants
	float I1 = traceFTransposeF(idx);
	float I3 = determinantFTransposeF(idx);

	//printf("I1 = %4.8f \n", I1);
	//printf("I3 = %4.8f \n", I3);

	//4. Calculate First Piola-Kirchoff Stress Tensor
	calculateFirstPiolaKirchoffTensor_NEO_HOOKEAN(idx, mu, lambda, I3);

	//printf("PF: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 3; ++col)
	//	{
	//		printf("%4.8f,", FirstPiolaKirchoffTensor[idx][row][col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");

	//5. Calculate StrainEnergy
	float strainEnergy = calculateStrainEnergy_NEO_HOOKEAN(volume[idx], lambda, mu, I1, I3);

	//printf("StrainEnergy = %4.8f \n", strainEnergy);

	//6. Calculate Strain Energy Gradient
	calculateStrainEnergyGradient_NEO_HOOKEAN(idx, volume[idx], refShapeMatrixInverse);

	//printf("Strain Energy Gradient: \n");
	//for (int row = 0; row < 3; ++row)
	//{
	//	for (int col = 0; col < 4; ++col)
	//	{
	//		printf("%4.8f,", Gradient[idx][row][col]);
	//	}
	//	printf("\n");
	//}
	//printf("\n \n");

	//7. Calculate Lagrange Multiplier
	float denominator = calculateLagrangeMultiplierDenominator(idx, inverseMass);

	if (denominator == 0.0f)
	{
		return;
	}

	float lagrangeMultiplier = -(strainEnergy / denominator);

	//printf("lagrangeMultiplier = %4.8f \n", lagrangeMultiplier);

	

	//8. Update Positions
	updatePositions(idx, lagrangeMultiplier, positions, inverseMass);
}


hipError_t projectConstraints(std::vector<int>& indices,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& positions_result,
	const CUDAPBD_SolverSettings& settings);

int CUDA_projectConstraints(std::vector<int> indices,
	std::vector<float> positions,
	std::vector<float> inverseMasses,
	std::vector<float> refShapeMatrixInverses,
	std::vector<float> volumes,
	const CUDAPBD_SolverSettings& settings)
{
	//GPU
	hipError_t cudaStatus = projectConstraints(indices, positions,
		inverseMasses, refShapeMatrixInverses, volumes, positions, settings);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Critical Error, aborting...");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	return 0;
}

hipError_t cudaErrorWrapper(hipError_t status)
{
	if (status != hipSuccess)
	{
		std::cerr << "ERROR: " << hipGetErrorString(status) << std::endl;
	}

	return status;
}

bool checkCudaErrorStatus(hipError_t status)
{
	if (status != hipSuccess)
	{
		std::cerr << "ERROR: " << hipGetErrorString(status) << std::endl;
		return false;
	}
	return true;
}

void getCudaDeviceProperties(int device)
{
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	std::cout << "Compute Capabilities for " << properties.name << " : " << std::endl;
	std::cout << "Major: " << properties.major << ", Minor: " << properties.minor << std::endl;
	std::cout << "Details: " << std::endl;
	std::cout << "	Num of SM    : " << properties.multiProcessorCount << std::endl;
	std::cout << "	Mem per Block: " << properties.sharedMemPerBlock << std::endl;
	std::cout << "	Mem per SM   : " << properties.sharedMemPerMultiprocessor << std::endl;
}

void queryCUDADevices()
{
	hipError_t deviceStatus;

	int deviceCount = 0;
	deviceStatus == hipGetDeviceCount(&deviceCount);

	std::cout << "Num CUDA Devices Found: " << deviceCount << std::endl;
	deviceStatus = cudaErrorWrapper(hipSetDevice(0));
	getCudaDeviceProperties(0);
}

hipError_t projectConstraints(std::vector<int>& indices,
	std::vector<float>& positions,
	std::vector<float>& inverseMasses,
	std::vector<float>& refShapeMatrixInverses,
	std::vector<float>& volumes,
	std::vector<float>& positions_result,
	const CUDAPBD_SolverSettings& settings)
{
	float* dev_positions;
	float* dev_inverseMasses;
	int* dev_indices;
	float* dev_refShapeMatrixInverses;
	float* dev_volumes;

	hipError_t deviceStatus;

	//Allocate memory
	deviceStatus = hipSetDevice(0);
	
	std::cout << "Allocating Memory..." << std::endl;

	std::cout << indices.size() << std::endl;
	std::cout << positions.size() << std::endl;
	std::cout << inverseMasses.size() << std::endl;
	std::cout << refShapeMatrixInverses.size() << std::endl;
	std::cout << volumes.size() << std::endl;

	deviceStatus = hipMalloc((void**)&dev_indices, indices.size() * sizeof(int));
	//checkCudaErrorStatus(deviceStatus);
	//std::cout << "1 Memory..." << std::endl;
	deviceStatus = hipMalloc((void**)&dev_positions, positions.size() * sizeof(float));
	//checkCudaErrorStatus(deviceStatus);
	//std::cout << "2 Memory..." << std::endl;
	deviceStatus = hipMalloc((void**)&dev_inverseMasses, inverseMasses.size() * sizeof(float));
	//checkCudaErrorStatus(deviceStatus);
	//std::cout << "3 Memory..." << std::endl;
	deviceStatus = hipMalloc((void**)&dev_refShapeMatrixInverses, refShapeMatrixInverses.size() * sizeof(float));
	//checkCudaErrorStatus(deviceStatus);
	//std::cout << "4 Memory..." << std::endl;
	deviceStatus = hipMalloc((void**)&dev_volumes, volumes.size() * sizeof(float));
	//checkCudaErrorStatus(deviceStatus);
	//std::cout << "5 Memory..." << std::endl;
	std::cout << "Copying Memory..." << std::endl;

	//Cpy memory
	deviceStatus = hipMemcpy(dev_indices, &indices[0], indices.size() * sizeof(int), hipMemcpyHostToDevice);
	//checkCudaErrorStatus(deviceStatus);
	deviceStatus = hipMemcpy(dev_positions, &positions[0], positions.size() * sizeof(float), hipMemcpyHostToDevice);
	//checkCudaErrorStatus(deviceStatus);
	deviceStatus = hipMemcpy(dev_inverseMasses, &inverseMasses[0], inverseMasses.size() * sizeof(float), hipMemcpyHostToDevice);
	//checkCudaErrorStatus(deviceStatus);
	deviceStatus = hipMemcpy(dev_refShapeMatrixInverses, &refShapeMatrixInverses[0], refShapeMatrixInverses.size() * sizeof(float), hipMemcpyHostToDevice);
	//checkCudaErrorStatus(deviceStatus);
	deviceStatus = hipMemcpy(dev_volumes, &volumes[0], volumes.size() * sizeof(float), hipMemcpyHostToDevice);

	//Execute Kernel
	std::cout << "Executing Kernel..." << std::endl;
	for (int it = 0; it < settings.numIterations; ++it)
	{
		solveFEMConstraint <<<settings.numBlocks, settings.numThreadsPerBlock>>>(
			dev_positions, dev_indices, dev_inverseMasses,
			dev_volumes, dev_refShapeMatrixInverses,
			settings.lambda, settings.mu, settings.trueNumberOfConstraints);

		cudaErrorWrapper(hipDeviceSynchronize());
	}
	std::cout << "Done..." << std::endl;


	//Cpy memory back
	positions_result.resize(positions.size());
	deviceStatus = cudaErrorWrapper(hipMemcpy(&positions_result[0], dev_positions, positions_result.size() * sizeof(float), hipMemcpyDeviceToHost));
	//checkCudaErrorStatus(deviceStatus);

	//Free memory
	hipFree(dev_positions);
	hipFree(dev_inverseMasses);
	hipFree(dev_indices);
	hipFree(dev_refShapeMatrixInverses);
	hipFree(dev_volumes);
	return deviceStatus;
}